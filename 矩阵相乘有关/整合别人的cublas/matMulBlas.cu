#include "hip/hip_runtime.h"
/*********************************************
该程序的功能是借用cublas来做矩阵并行运算,C=AB
考虑到cublas中矩阵是按列存储的
调用hipblasSgemm时是按C=BA调用,实际执行的是B^T x A^T ,由于按列存储，这样乘积得到的矩阵刚好是按行存储的C，打印结果是就很方便了
因此实际调用该程序时，矩阵A和矩阵B按行存储，结果矩阵C也是按行存储的，同C语言矩阵乘法一样。
*********************************************/
#include<hipblas.h>
#include<hipblas.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <hiprand.h>
#include <iostream>
#include<fstream>
#include<vector>
#include<string>
#include "MyTime.h"
#define A_ROW 15000      //随意定义A和B的size
#define A_COL 15000
#define B_COL 15000
#define CUDA_SAFE_CALL(cuda_errno) { \
if(hipSuccess!=cuda_errno) {\
			printf("\nCUDA Runtime Error:%s(err_num=%d)\n", hipGetErrorString(cuda_errno),cuda_errno);\
			printf("\nCUDA Error in%s,line %d\n",__FILE__,__LINE__);\
			hipDeviceReset();\
			hipSetDevice(0);\
}}
using namespace std;
extern void gpu_blas_mmul(hipblasHandle_t &handle, float *C, const float *A, const float *B, const int m, const int k, const int n);
extern void save_to_text(std::string filename, std::vector<double> &p);
int main()
{
	
	//分配cpu上的内存
	float *host_A=(float*) malloc(sizeof(float)*A_ROW*A_COL);
	float *host_B=(float*) malloc(sizeof(float)*A_COL*B_COL);
	float *host_C=(float*) malloc(sizeof(float)*A_ROW*B_COL);
	
	//GPU上内存分配
	float *dev_A=0,*dev_B=0,*dev_C=0;
	MyTimer t;
	hipMalloc((void**)&dev_A,sizeof(float)* A_ROW* A_COL );
	hipMalloc((void**)&dev_B,sizeof(float)* A_COL* B_COL );
	hipMalloc((void**)&dev_C,sizeof(float)* A_ROW* B_COL );
	
	
	hipblasHandle_t handle;//handle，表示是cublas的上下文句柄,初始化cublas
	hipblasCreate(&handle);//计算前create handle
	vector<double> mytime;
	int my_ROW;
	for(int i=0;i<100;i++)
	{
	my_ROW=150*(i+1);
	t.start();//计算时用time.h库中的clock()函数
 // 将矩阵数据传递进 显存 中已经开辟好了的空间
    hipblasSetVector (
        my_ROW*A_COL,    // 要存入显存的元素个数
        sizeof(float),    // 每个元素大小
        host_A,    // 主机端起始地址
        1,    // 连续元素之间的存储间隔
        dev_A,    // GPU 端起始地址
        1    // 连续元素之间的存储间隔
    );
	hipblasSetVector (
        A_COL*B_COL,    // 要存入显存的元素个数
        sizeof(float),    // 每个元素大小
        host_B,    // 主机端起始地址
        1,    // 连续元素之间的存储间隔
        dev_B,    // GPU 端起始地址
        1    // 连续元素之间的存储间隔
    );
	
	gpu_blas_mmul(handle,dev_C,dev_A, dev_B, my_ROW,B_COL, A_COL);
	hipDeviceSynchronize();
	// 从 显存 中取出运算结果至 内存中去
    hipblasGetVector (
        my_ROW*B_COL,    //  要取出元素的个数
        sizeof(float),    // 每个元素大小
        dev_C,    // GPU 端起始地址
        1,    // 连续元素之间的存储间隔
        host_C,    // 主机端起始地址
        1    // 连续元素之间的存储间隔
    );
	t.stop();
  	mytime.push_back(t.elapse());
	}
	
	save_to_text("CUDATime.txt", mytime);
	
	hipFree(dev_A);
	hipFree(dev_B);
	hipFree(dev_C);
	hipblasDestroy(handle); 

	free(host_A);
	free(host_B);
	free(host_C);
	return 0;
}
//计算C=A*B，m,n分别是C的行和列，k是A的行和B的列,但是考虑到cublas是按列存储，
//hipblasSgemm的参数，把自己绕的有点晕。这里用的的是B*A，即（A_COL*B_COL）*（A_ROW*A_COL）
//但是真正在函数中执行的是BT*AT=CT(即转置相乘，即B_COL*A_COL）*（A_COL*A_ROW））从而不用自己在麻烦机器来转置了.lda,ldb分别对应B_COL，A_COLB_COL，A_COL即对应的是实际执行的矩阵的row，ldc对应的
void gpu_blas_mmul(hipblasHandle_t &handle, float *C,const float *A, const float *B,  const int m,const int n, const int k ) 
{
	const float alpha = 1.0;
	const float beta = 0.0;
	hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,n,m,k, &alpha,B,n,A,k,&beta,C,n);
}
void save_to_text(std::string filename, std::vector<double> &p)
{
	using namespace std;
	fstream outfile;
	string tempfilenanme = filename;
	outfile.open(tempfilenanme, ios::out | ios::trunc);
	for (auto i = 0; i < p.size(); i++)
		outfile << p[i] << "\r"<<endl;
	outfile.close();
}	


